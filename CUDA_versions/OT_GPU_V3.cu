#include <iostream>
#include <vector>
#include <cmath>
#include <cstdint>
#include <fstream>
#include <hip/hip_runtime.h>

#include <iostream>
#include <chrono>
#include <ctime>
#include <cmath>

#define H 720
#define W 1280
#define C 3

#define R_lower 230
#define R_upper 255
#define G_lower 0
#define G_upper 15
#define B_lower 0
#define B_upper 15

//er word voor iedere pixel een thread voorzien => 720 * 1280 = 921.600 threads = overklokken scenario => threads moeten in groepjes sequentieel uitvoeren 


__global__ void find_object_colour(const uint8_t* pixels, int* row_counts, int* col_counts) {
    //doordat met array werkt en er per element van elk van de arrays
    //slechts 1 thread moet aanpassingen doen is geen atomic operation nodig

    int i = blockIdx.x * blockDim.x + threadIdx.x; //thread index

    if(i < W * H){
        int idx = i * C; //index waarnaar thread i moet kijken in de flattened image array
                         //  0        1        2        => thread index (i)  |
                         // [R, G, B, R, G, B, R, G, B]                      | => i * C (= 3) = idx
                         //  0  1  2  3  4  5  6  7  8  => array index (idx) |

        //printf("idx: %d \n", idx);

        int count = 0;

        uint8_t r = pixels[idx];
        uint8_t g = pixels[idx + 1];
        uint8_t b = pixels[idx + 2];

        if (r >= R_lower && r <= R_upper &&
            g >= G_lower && g <= G_upper &&
            b >= B_lower && b <= B_upper) {

            int row_index = i / W;
            int column_index = i % W;
            
            //count++; 
            atomicAdd(&row_counts[row_index], 1); //we moeten hier een atomicAdd doen aangezien iedere thread nu één pixel vertegenwoordigt
                                                  //=> meerdere threads accessen gelijktijdig dezelfde index in de row_counts array en de
                                                  //col_counts array aangezien meerdere pixels op eenzelfde rij en kolom staan (deze arrays 
                                                  //hebben een lengte die gelijk is aan het aantal kolommen en rijen in de afbeelding)
                                                  //
                                                  //mogelijke andere manier:
                                                  //één flattened array maken met grote 921600 (voor iedere pixel een slot) en dan in preprocessing
                                                  //de index in deze flat array terugrekenen naar een matrix en zo het gemiddelde bepalen door het
                                                  //midden te tellen op de CPU te doen
            atomicAdd(&col_counts[column_index], 1);  
        }

        //int row_index = i / W; //    0  1  2
                                 // 0 [R, G, B]
                                 // 1 [X, G, B]      0  1  2    3  4  5    6  7  8
                                 // 2 [R, G, B] => [[R, G, B], [X, G, B], [R, G, B]] => X [3 = idx] (array) => 3 / 3 (= W) = 1 = row_index (matrix)
        //printf("row_index: %d \n", row_index);

        //int column_index = i % W; //rest geeft kolom weer
        //printf("column_index: %d \n", column_index);

        // row_counts[row_index] = count;
        // col_counts[column_index] = count;
    }
}

uint8_t* get_image_array_ppm(const char* image) {
    FILE* imageFile = fopen(image, "rb");
    if (!imageFile) {
        perror("ERROR: Cannot open image file");
        exit(EXIT_FAILURE);
    }

    char header[2];
    fscanf(imageFile, "%s", header);
    int width, height, maxval;
    fscanf(imageFile, "%d %d", &width, &height);
    fscanf(imageFile, "%d", &maxval);
    fgetc(imageFile);

    uint8_t* image_array = (uint8_t*)malloc(H * W * C);
    fread(image_array, sizeof(uint8_t), H * W * C, imageFile);
    fclose(imageFile);
    return image_array;
}

void save_coordinates_to_file(const std::vector<std::pair<int, int>>& coordinates) {
    std::ofstream outFile("./output/coordinates_cuda_V3.txt");
    for (const auto& p : coordinates) {
        outFile << p.first << " " << p.second << "\n";
    }
    outFile.close();
}

int main() {
    std::vector<std::pair<int, int>> object_coordinates;

    std::vector<float> kernel_times;

    uint8_t* d_pixels;
    int* d_row_counts;
    int* d_col_counts;

    //std::chrono::time_point<std::chrono::system_clock> StartTime = std::chrono::system_clock::now();

    for (int frameNr = 0; frameNr < 150; ++frameNr) {
        std::string path = "./CUDA_versions/frames/moving_ball_frames_ppm/" + std::to_string(frameNr) + ".ppm";
        uint8_t* pixels = get_image_array_ppm(path.c_str());

        std::vector<int> row_counts(H, 0);
        std::vector<int> col_counts(W, 0);

        hipMalloc(&d_pixels, H * W * C);
        hipMemcpy(d_pixels, pixels, H * W * C, hipMemcpyHostToDevice);

        hipMalloc(&d_row_counts, H * sizeof(int));
        hipMalloc(&d_col_counts, W * sizeof(int));
        hipMemset(d_row_counts, 0, H * sizeof(int));
        hipMemset(d_col_counts, 0, W * sizeof(int));

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // kernel gegevens
        int row_block_size = 1024;
        int col_block_size = 1024;
        int grid_size = ((H * W) / row_block_size); //levert 900 blocks op

        hipEventRecord(start);
        find_object_colour<<<grid_size, row_block_size>>>(d_pixels, d_row_counts, d_col_counts); //wordt gecalled met 900 blocks van elk 1024 threads
        hipEventRecord(stop);                                                     
        hipDeviceSynchronize();

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        kernel_times.push_back(milliseconds);

        hipMemcpy(row_counts.data(), d_row_counts, H * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(col_counts.data(), d_col_counts, W * sizeof(int), hipMemcpyDeviceToHost);

        //Dit wnr reductie is ook op GPU doen, nu op CPU
        //kolom/rijnummer in array zetten en op een manier midden zoeken
        double row_sum = 0, row_count = 0, col_sum = 0, col_count = 0;
        for (int i = 0; i < H; ++i) {
            if (row_counts[i]) { row_sum += i; row_count += 1; }
        }
        for (int j = 0; j < W; ++j) {
            if (col_counts[j]) { col_sum += j; col_count += 1; }
        }

        int row_mean = (row_count > 0) ? round(row_sum / row_count) : 0;
        int col_mean = (col_count > 0) ? round(col_sum / col_count) : 0;
        object_coordinates.push_back({row_mean, col_mean});

        free(pixels);
        hipFree(d_pixels);
        hipFree(d_row_counts);
        hipFree(d_col_counts);

        std::cout << milliseconds << " milliseconds" <<std::endl;
    }

    // std::chrono::time_point<std::chrono::system_clock> EndTime = std::chrono::system_clock::now();

    // int milliseconds = std::chrono::duration_cast<std::chrono::milliseconds>(EndTime - StartTime).count();

    // std::cout << "Time: " << milliseconds << " milliseconds.";

    float total_time = 0.0f;
    for (float t : kernel_times) total_time += t;
    float average_time = (kernel_times.size() > 0) ? total_time / kernel_times.size() : 0.0f;

    std::cout << "Total kernel execution time: " << total_time << " ms" << std::endl;
    std::cout << "Average kernel time per frame: " << average_time << " ms" << std::endl;


    save_coordinates_to_file(object_coordinates) ;
    return 0;
}
