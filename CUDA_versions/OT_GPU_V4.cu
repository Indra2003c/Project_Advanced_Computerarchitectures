#include <iostream>
#include <vector>
#include <cmath>
#include <cstdint>
#include <fstream>
#include <hip/hip_runtime.h>

#include <iostream>
#include <chrono>
#include <ctime>
#include <cmath>

#define H 720
#define W 1280
#define C 3

#define R_lower 230
#define R_upper 255
#define G_lower 0
#define G_upper 15
#define B_lower 0
#define B_upper 15

//er word voor iedere kleurcomponent een thread voorzien => 720 * 1280 * 3 = 2.764.800 threads = extreme overklokken scenario => threads moeten in groepjes sequentieel uitvoeren
//merk ook op dat we hier wel gebruik kunnen maken van coalesced memory: alle kleurcomponenten zitten naast elkaar in het geheugen en opeenvolgende threads accessen opeenvolgende
//kleurcomponenten


__global__ void find_object_colour(const uint8_t* pixels, int* row_counts, int* col_counts) {
    //doordat met array werkt en er per element van elk van de arrays
    //slechts 1 thread moet aanpassingen doen is geen atomic operation nodig

    int i = blockIdx.x * blockDim.x + threadIdx.x; //thread index

    int count = 0;

    int row_index = -1;
    int column_index = -1;

    if(i % 3 == 0) { //R component
        uint8_t r = pixels[i];
        if(r >= R_lower && r <= R_upper) {
            count++;
        }
        row_index = (i / 3) / W;
        column_index = (i / 3) % W;
    }else if(i % 3 == 1) { //G component
        uint8_t g = pixels[i];
        if(g >= G_lower && g <= G_upper) {
            count++;
        }
        row_index = ((i / 3) - 1) / W; //i - 1 om de index terug te herleiden naar de index van de R component zodat dit dan omgerekend
                               //kan worden naar de pixel index (rij + kolom) in de foto (zoals in V3)
        column_index = ((i / 3) - 1) % W;
    }else { //B component
        uint8_t b = pixels[i];
        if(b >= B_lower && b <= B_upper) {
            count++;
        }
        row_index = ((i / 3) - 2) / W;
        column_index = ((i / 3) - 2) % W;
    }

    row_counts[row_index] = count;
    col_counts[column_index] = count;
}

uint8_t* get_image_array_ppm(const char* image) {
    FILE* imageFile = fopen(image, "rb");
    if (!imageFile) {
        perror("ERROR: Cannot open image file");
        exit(EXIT_FAILURE);
    }

    char header[2];
    fscanf(imageFile, "%s", header);
    int width, height, maxval;
    fscanf(imageFile, "%d %d", &width, &height);
    fscanf(imageFile, "%d", &maxval);
    fgetc(imageFile);

    uint8_t* image_array = (uint8_t*)malloc(H * W * C);
    fread(image_array, sizeof(uint8_t), H * W * C, imageFile);
    fclose(imageFile);
    return image_array;
}

void save_coordinates_to_file(const std::vector<std::pair<int, int>>& coordinates) {
    std::ofstream outFile("./output/coordinates_cuda_V4.txt");
    for (const auto& p : coordinates) {
        outFile << p.first << " " << p.second << "\n";
    }
    outFile.close();
}

int main() {
    std::vector<std::pair<int, int>> object_coordinates;

    

    uint8_t* d_pixels;
    int* d_row_counts;
    int* d_col_counts;

    std::chrono::time_point<std::chrono::system_clock> StartTime = std::chrono::system_clock::now();

    for (int frameNr = 0; frameNr < 150; ++frameNr) {
        std::string path = "./CUDA_versions/frames/moving_ball_frames_ppm/" + std::to_string(frameNr) + ".ppm";
        uint8_t* pixels = get_image_array_ppm(path.c_str());

        std::vector<int> row_counts(H, 0);
        std::vector<int> col_counts(W, 0);

        hipMalloc(&d_pixels, H * W * C);
        hipMemcpy(d_pixels, pixels, H * W * C, hipMemcpyHostToDevice);

        hipMalloc(&d_row_counts, H * sizeof(int));
        hipMalloc(&d_col_counts, W * sizeof(int));
        hipMemset(d_row_counts, 0, H * sizeof(int));
        hipMemset(d_col_counts, 0, W * sizeof(int));

        // kernel gegevens
        int row_block_size = 1024;
        int col_block_size = 1024;
        int grid_size = ((H * W * C) / row_block_size); //levert 2700 blocks op

        find_object_colour<<<grid_size, row_block_size>>>(d_pixels, d_row_counts, d_col_counts); //wordt gecalled met 900 blocks van elk 1024 threads                                                       
        hipDeviceSynchronize();

        hipMemcpy(row_counts.data(), d_row_counts, H * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(col_counts.data(), d_col_counts, W * sizeof(int), hipMemcpyDeviceToHost);

        //Dit wnr reductie is ook op GPU doen, nu op CPU
        //kolom/rijnummer in array zetten en op een manier midden zoeken
        double row_sum = 0, row_count = 0, col_sum = 0, col_count = 0;
        for (int i = 0; i < H; ++i) {
            if (row_counts[i]) { row_sum += i; row_count += 1; }
        }
        for (int j = 0; j < W; ++j) {
            if (col_counts[j]) { col_sum += j; col_count += 1; }
        }

        int row_mean = (row_count > 0) ? round(row_sum / row_count) : 0;
        int col_mean = (col_count > 0) ? round(col_sum / col_count) : 0;
        object_coordinates.push_back({row_mean, col_mean});

        free(pixels);
        hipFree(d_pixels);
        hipFree(d_row_counts);
        hipFree(d_col_counts);
    }

    std::chrono::time_point<std::chrono::system_clock> EndTime = std::chrono::system_clock::now();

    int milliseconds = std::chrono::duration_cast<std::chrono::milliseconds>(EndTime - StartTime).count();

    std::cout << "Time: " << milliseconds << " milliseconds.";

    save_coordinates_to_file(object_coordinates) ;
    return 0;
}
